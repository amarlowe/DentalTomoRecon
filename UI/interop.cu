#include "hip/hip_runtime.h"
#include "interop.h"

//#include <helper_gl.h>
#include <GL/glew.h>
#include <GL/freeglut.h>

// includes, cuda
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#define MAX(a,b) ((a > b) ? a : b)

__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width + x] = make_float4(u, w, v, 1.0f);
}

interop::interop(int *argc, char **argv, int x, int y, bool first) {

	width = x;
	height = y;
	
	if (first) glutInit(argc, argv);
	glewInit();
	//glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	//glutInitWindowSize(x, y);

	glClearColor(0.0, 0.0, 0.0, 0.0);
	glDisable(GL_DEPTH_TEST);
	glClear(GL_COLOR_BUFFER_BIT);
	glViewport(0, 0, (GLint)x, (GLint)y);

	// projection
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)x / (GLfloat)y, 0.1, 10.0);

	SDK_CHECK_ERROR_GL();

	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());

	// create VBO
	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	// run the cuda part
	// map OpenGL buffer object for writing from CUDA
	float4 *dptr;
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
		cuda_vbo_resource));

	launch_kernel(dptr, (GLfloat)x / 2, (GLfloat)y / 2, g_fAnim);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));

	// start rendering mainloop
	//glutMainLoop();
}

interop::~interop() {
	if (vbo)
		deleteVBO(&vbo, cuda_vbo_resource);
}

void interop::createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res, unsigned int vbo_res_flags) {
	assert(vbo);

	// create buffer object
	glGenBuffers(1, vbo);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);

	// initialize buffer object
	unsigned int size = width * height * sizeof(float);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

	SDK_CHECK_ERROR_GL();
}

void interop::deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res){
	// unregister this buffer object with CUDA
	checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}

void interop::display(int x, int y)
{
	width = x;
	height = y;
	glViewport(0, 0, (GLint)x, (GLint)y);

	// run CUDA kernel to generate vertex positions
	runCuda(&cuda_vbo_resource);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 1.0, 0.0);

	// render from the vbo
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(4, GL_FLOAT, 0, 0);

	glEnableClientState(GL_VERTEX_ARRAY);
	glColor3f(1.0, 0.0, 0.0);
	glDrawArrays(GL_POINTS, 0, width * height / 4);
	glDisableClientState(GL_VERTEX_ARRAY);

	//glutSwapBuffers();

	g_fAnim += 0.01f;

	//computeFPS();
}

void interop::launch_kernel(float4 *pos, unsigned int mesh_width, unsigned int mesh_height, float time){
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	simple_vbo_kernel <<< grid, block >>>(pos, mesh_width, mesh_height, time);
}

void interop::runCuda(struct hipGraphicsResource **vbo_resource)
{
	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

	// map OpenGL buffer object for writing from CUDA
	float4 *dptr;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
		*vbo_resource));
	//printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

	// execute the kernel
	//    dim3 block(8, 8, 1);
	//    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	//    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

	launch_kernel(dptr, width/2, height/2, g_fAnim);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}


